#include "hip/hip_runtime.h"
#include "mex.h"
#include "wrapperMx.h"
#include "_maxpool3d_gpu.h"


namespace {

//// 
const int   NUM_THDS = 512;

mwSize ceil_divide (mwSize a, mwSize b) {
  return (a + b - 1)/b;
}

//// thin wrappers 
struct tw_array5d { 
  float *beg;
  mwSize sz[5];
  mwSize HW, HWD;
  mwSize nelem;
};

template<typename T>
struct tw_vec {
  T* beg;
  mwSize sz;
};

struct fprop_impl {
  tw_array5d     X, Y;
  tw_vec<double> ind;
 
  mwSize pool[3];
  mwSize stride[3];
  mwSize pad[6];
};

struct bprop_impl
{
  tw_vec<float>  dX, dY;
  tw_vec<double> ind;
};

//// kernel Impl
__device__ const float VERY_NEGATIVE_NUM = -1e20;

__device__ void ind2sub (mwSize iElem, mwSize sz[5], 
                         mwSize &h, mwSize &w, mwSize &d, mwSize &iVol) 
{
  mwSize H   = sz[0];
  mwSize HW  = H * sz[1];
  mwSize HWD = HW * sz[2];

  iVol = iElem / HWD;
  iElem = iElem % HWD;

  d = iElem / HW;
  iElem = iElem % HW;

  w = iElem / H;

  h = iElem % H;
}

__global__ void kernel_fprop (fprop_impl impl) {
  mwSize iElem = blockIdx.x * blockDim.x + threadIdx.x;

  if (iElem >= impl.Y.nelem) return;

  // subscript on Y
  mwSize iY, jY, kY, iVol;
  ind2sub(iElem, impl.Y.sz,  iY,jY,kY,iVol);

  // init value for current Y
  float  vmax = VERY_NEGATIVE_NUM;
  double imax = -43.0;

  // set the window on X for current Y element (iElem); note the offset can be negative
  mwSize xH   = impl.X.sz[0];
  mwSize xHW  = impl.X.HW;
  mwSize xHWD = impl.X.HWD;
  int64_T xwin_offset[3];
  xwin_offset[0] = -static_cast<int64_T>( impl.pad[0]) + 
                    static_cast<int64_T>( iY * impl.stride[0] ); 
  xwin_offset[1] = -static_cast<int64_T>( impl.pad[2]) + 
                    static_cast<int64_T>( jY * impl.stride[1] );
  xwin_offset[2] = -static_cast<int64_T>( impl.pad[4] ) + 
                    static_cast<int64_T>( kY * impl.stride[2] );
  const float* const xwin_beg = impl.X.beg + 
                                xwin_offset[0] + 
                                xwin_offset[1]*xH + 
                                xwin_offset[2]*xHW +
                                iVol*xHWD;

  // inspect the window at X, get the max value
  for (int64_T t = 0; t < impl.pool[2]; ++t) {     // X window dim3: depth
    int64_T xt = t + xwin_offset[2];
    bool xtInRange = (xt>=0) && (xt<impl.X.sz[2]);

    for (int64_T s = 0; s < impl.pool[1]; ++s) {   // X window dim2: width
      int64_T xs = s + xwin_offset[1];
      bool xsInRange = (xs>=0) && (xs<impl.X.sz[1]);

      for (int64_T r = 0; r < impl.pool[0]; ++r) { // X window dim1: height
        int64_T xr = r + xwin_offset[0];
        bool xrInRange = (xr>=0) && (xr<impl.X.sz[0]);

        // if out of range: never collect the element
        if ( !(xtInRange && xsInRange && xrInRange) )
          continue;

        // collect the element: current x value
        float vx = *(xwin_beg + r + s*xH + t*xHW);
        if (vx >= vmax) { // found new max value?
          vmax = vx;
          imax = double( xr + xs*xH + xt*xHW + iVol*xHWD );
        } // if

      } // r
    } // s
  } // t

  // write to the target
  impl.Y.beg[iElem]   = vmax;
  impl.ind.beg[iElem] = imax + 1; // to Matlab 1-base
}

__global__ void kernel_bprop (bprop_impl impl) {
  mwSize iY = blockIdx.x * blockDim.x + threadIdx.x;

  if (iY >= impl.dY.sz) return;

  mwSize ix = mwSize( impl.ind.beg[iY] );
  ix -= 1;

  // atomic Increment: there can be overlapping ix!
  atomicAdd( (impl.dX.beg + ix), impl.dY.beg[iY] );
}

} // namespace

//// impl of public methods
maxpool3d_gpu::maxpool3d_gpu()
{

}

maxpool3d_gpu::maxpool3d_gpu(const maxpool3d &obj)
{
  for (int i = 0; i < 6; ++i) pad[i]  = obj.pad[i];
  for (int i = 0; i < 3; ++i) pool[i] = obj.pool[i];
  for (int i = 0; i < 3; ++i) stride[i] = obj.stride[i];

  ind = obj.ind;
  X  = obj.X;
  dX = obj.dX;
  Y  = obj.Y;
  dY = obj.dY;

  ct = obj.ct;

}

void maxpool3d_gpu::fprop()
{
  // create output
  create_Y();
  create_ind();


  // set the impl struct and run it
  fprop_impl impl;
  // options
  for (int i = 0; i < 3; ++i) impl.pool[i] = pool[i];
  for (int i = 0; i < 6; ++i) impl.pad[i] = pad[i];
  for (int i = 0; i < 3; ++i) impl.stride[i] = stride[i];
  // input: X, device pointer
  impl.X.beg = (float*) X.getDataBeg();
  for (int i = 0; i < 5; ++i) impl.X.sz[i] = X.getSizeAtDim(i);
  impl.X.HW    = impl.X.sz[0] * impl.X.sz[1];
  impl.X.HWD   = impl.X.HW * impl.X.sz[2];
  impl.X.nelem = numel(X);
  // output: Y, device pointer
  impl.Y.beg = (float*) Y.getDataBeg();
  for (int i = 0; i < 5; ++i) impl.Y.sz[i] = Y.getSizeAtDim(i);
  impl.Y.HW    = impl.Y.sz[0] * impl.Y.sz[1];
  impl.Y.HWD   = impl.Y.HW * impl.Y.sz[2];
  impl.Y.nelem = numel(Y);
  // output: ind, device pointer
  impl.ind.beg = (double *) ind.getDataBeg();
  impl.ind.sz  = numel(ind);


  // run
  mwSize nelem = numel(Y);
  kernel_fprop<<<ceil_divide(nelem, NUM_THDS), NUM_THDS>>>( impl );
}

void maxpool3d_gpu::bprop()
{
  // create dX at input port
  check_dY_ind();
  create_dX();


  // set the impl struct
  bprop_impl impl;
  //
  impl.dX.beg = (float*) dX.getDataBeg();
  impl.dX.sz  = numel(dX);
  //
  impl.dY.beg = (float*) dY.getDataBeg();
  impl.dY.sz  = numel(dY);
  //
  impl.ind.beg = (double*) ind.getDataBeg();
  impl.ind.sz  = numel(ind);


  // run
  kernel_bprop <<<ceil_divide(impl.dY.sz, NUM_THDS), NUM_THDS>>>( impl );
}

