#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "_conv3d_blas_gpu.h"

namespace {
//// helpers for threads
mwSize ceil_divide (mwSize a, mwSize b) {
  return (a + b - 1)/b;
}

const int NUM_THD_DIM = 512; 

//// helper: setting initial value
template<typename T>
__global__ void kernelSetZero (T* beg, mwSize len) {
  mwSize ind = blockIdx.x * blockDim.x + threadIdx.x;
  if (ind < len) beg[ind] = static_cast<T>(0);
}

template<typename T>
__global__ void kernelSetOne (T* beg, mwSize len) {
  mwSize ind = blockIdx.x * blockDim.x + threadIdx.x;
  if (ind < len) beg[ind] = static_cast<T>(1);
}

//// Impl of copying data back and forth for Vol and Convmat
typedef conv3d_blas_gpu::CpyVolConvmatImpl CpyImpl;

__device__ mwSize get_convmat_h (CpyImpl &ip, mwSize indCM) {
  return (indCM % ip.convmat.H);
}

__device__ mwSize get_convmat_w (CpyImpl &ip, mwSize indCM) {
  return (indCM / ip.convmat.H);
}

__device__ void get_subY (CpyImpl &ip, mwSize ind,  mwSize subY[3]) {
  mwSize HW = ip.szY[0] * ip.szY[1];
  mwSize H  = ip.szY[0];

  subY[2] = ind / HW;
  ind %= HW;

  subY[1] = ind / H;
  ind %= H;

  subY[0] = ind;
}

__device__ void get_win_offset4 (CpyImpl &ip, mwSize h_covnmat,  int64_T win_offset[3]) {
  mwSize subY[3];
  get_subY(ip, h_covnmat, subY);

  for (int i = 0; i < 3; ++i) 
    win_offset[i] = -static_cast<int64_T>(ip.pad[2*i]) + static_cast<int64_T>(subY[i] * ip.stride[i]);
  win_offset[3] = 0;
}

__device__ void get_win_sub4 (CpyImpl &ip, mwSize w_convmat,  mwSize win_sub[4]) {
  mwSize H   = ip.szF[0]; 
  mwSize HW  = H * ip.szF[1];
  mwSize HWD = HW * ip.szF[2];

  win_sub[3] = w_convmat / HWD;
  w_convmat %= HWD;

  win_sub[2] = w_convmat / HW;
  w_convmat %= HW;

  win_sub[1] = w_convmat / H;
  w_convmat %= H;

  win_sub[0] = w_convmat;
}

// return -1 if out of range (either underflow or overflow)
__device__ int64_T get_indVol (CpyImpl &ip, int64_T win_offset[4], mwSize win_sub[4]) {

  // the global subscript and guaranteed valid range
  int64_T vol_sub[4];
  for (int i = 0; i < 4; ++i) {
    vol_sub[i] = win_offset[i] + static_cast<int64_T>(win_sub[i]);
    if ( vol_sub[i] < 0 ) return -1; // underflow
    if ( vol_sub[i] >= ip.vol_i.sz[i] ) return -1; // overflow
  }

  mwSize H   = ip.vol_i.sz[0];
  mwSize HW  = H * ip.vol_i.sz[1];
  mwSize HWD = HW * ip.vol_i.sz[2];

  return static_cast<int64_T>(HWD*vol_sub[3] + HW*vol_sub[2] + H*vol_sub[1] + vol_sub[0]);
}

const int DIR_VOL_TO_CONVMAT   = 0; // nvcc does not support enum instantiation?
const int DIR_VOL_FROM_CONVMAT = 1;

template<int dir>
void __global__ kernelCpyVolConvmat (CpyImpl ip) {
  mwSize indCM = blockDim.x * blockIdx.x + threadIdx.x;
  if ( indCM >= (ip.convmat.H*ip.convmat.W) ) return;

  // fill h, w
  mwSize h = get_convmat_h(ip, indCM); // convmat dim1
  mwSize w = get_convmat_w(ip, indCM); // convmat dim2

  // h (convmat dim1) -> window's offset (starting point) on volume (win_offset[3] = 0 as volume dim4 all in!)
  int64_T win_offset[4]; // fill win_offset
  get_win_offset4(ip, h, win_offset);

  // w (convmat dim2) -> win_sub ( r,s,t,u the subscript within the window )
  mwSize win_sub[4]; // (r, s, t, u) 
  get_win_sub4(ip, w, win_sub);

  // win_offset[4] and win_sub[4] -> linear index, ind, on volume
  int64_T indVol = get_indVol(ip, win_offset, win_sub);
  
  // copy the data at indCM, indVol
  if (indVol < 0) {
    if (dir == DIR_VOL_TO_CONVMAT) 
      ip.convmat.beg[indCM] = 0.0; // pad zeros!
    //else: DIR_VOL_FROM_CONVMAT, do nothing
    return;
  }
  
  if (dir == DIR_VOL_TO_CONVMAT) // vol -> convmat
    ip.convmat.beg[indCM] = ip.vol_i.beg[indVol];
  else { // DIR_VOL_FROM_CONVMAT, vol <- convmat
    // ATOMIC increment: ip.vol_i.beg[indVol] += ip.convmat.beg[indCM]
    atomicAdd( (ip.vol_i.beg + indVol), ip.convmat.beg[indCM]);
  }
}

} // namespace


//// impl of public methods
conv3d_blas_gpu::conv3d_blas_gpu()
{

}

conv3d_blas_gpu::conv3d_blas_gpu(const conv3d& obj)
{
  for (int i = 0; i < 6; ++i) pad[i]  = obj.pad[i];
  for (int i = 0; i < 3; ++i) stride[i] = obj.stride[i];

  X  = obj.X;
  dX = obj.dX;
  Y  = obj.Y;
  dY = obj.dY;
  F  = obj.F;
  dF = obj.dF;
  B  = obj.B;
  dB = obj.dB;

  ct = obj.ct;
}

void conv3d_blas_gpu::fprop()
{
  create_Y();
  init_convmat();
  init_u(); 

  try {
    // iterate over each training instance
    CpyVolConvmatImpl ip = make_initial_CpyVolConvmatImpl( X );
    mwSize N = X.getSizeAtDim(4);
    for (mwSize i = 0; i < N; i++) {
      // make phiX: the convolution matrix
      vol_to_convmat(ip, X, i);

      // convolution: Y_ = phiX * F_
      matw F_ = make_F_();
      matw Y_ = make_Y_(i);
      cu_AxBtoC(convmat, F_, Y_, true); // overwrite Y_ 

      // plus the bias: Y_ += u * B
      matw B_ = make_B_();
      cu_AxBtoC(u, B_, Y_, false); // accumulation on Y_
    } // for i
  } // try
  catch (const blas_ex& e) {
    free_u();
    free_convmat();
    throw conv3d_ex(e.what());
  }

  free_u();
  free_convmat();
}

void conv3d_blas_gpu::bprop()
{
  check_X_size();
  create_dX();
  create_dF();
  create_dB();
  init_convmat();
  init_u();

  try {
    // iterate over each instance
    CpyVolConvmatImpl ip = make_initial_CpyVolConvmatImpl( X );
    matw dF_ = make_dF_();
    matw dB_ = make_dB_();
    mwSize N = X.getSizeAtDim(4);
    for (mwSize i = 0; i < N; ++i) {
      // make phiX: the convolution matrix
      vol_to_convmat(ip, X, i);

      // dF += phiX' * dY_
      matw dY_ = make_dY_(i);
      cu_ATxBtoC(convmat, dY_, dF_, false); // accumulation on dF_ TODO: the right cublas

      // dB += u' * dY
      cu_ATxBtoC(u, dY_, dB_, false); // accumulation on dB_

      // dphiX = dY * F'
      matw F_ = make_F_();
      // safe to reuse convmat memory as X and dX have the same size; remember to overwrite it!
      cu_AxBTtoC(dY_, F_, convmat, true);
      // dX(:,:,:,:,i) <-- dphiX
      vol_from_convmat(ip, dX, i);
    }
  }
  catch (const blas_ex& e) {
    free_u();
    free_convmat();
    throw conv3d_ex(e.what());
  }

  free_u();
  free_convmat();
}

//// Impl of helper: fprop
matw conv3d_blas_gpu::make_F_()
{
  matw F_;
  F_.beg = (float*)F.getDataBeg();
  F_.H   = numelVol(F) * F.getSizeAtDim(3);
  F_.W   = F.getSizeAtDim(4);

  return F_;
}

matw conv3d_blas_gpu::make_Y_(mwSize i)
{
  matw Y_;
  Y_.beg = getVolInstDataBeg<float>(Y, i);
  Y_.H   = numelVol(Y);
  Y_.W   = Y.getSizeAtDim(3);

  return Y_;
}

matw conv3d_blas_gpu::make_B_()
{
  matw B_;
  B_.beg = (float*)B.getDataBeg();
  B_.H   = 1;
  B_.W   = numel(B);

  return B_;
}

//// Impl of helper: bprop
matw conv3d_blas_gpu::make_dY_(mwSize i)
{
  matw dY_;
  dY_.beg = getVolInstDataBeg<float>(dY, i);
  dY_.H   = numelVol(dY);
  dY_.W   = dY.getSizeAtDim(3);

  return dY_;
}

matw conv3d_blas_gpu::make_dF_()
{
  matw dF_;
  dF_.beg = (float*)dF.getDataBeg();
  dF_.H   = numelVol(dF) * dF.getSizeAtDim(3);
  dF_.W   = dF.getSizeAtDim(4);

  return dF_;
}

matw conv3d_blas_gpu::make_dB_()
{
  matw dB_;
  dB_.beg = (float*)dB.getDataBeg();
  dB_.H   = 1;
  dB_.W   = numel(dB);
  
  return dB_;
}

//// Impl of helper: the stacked matrix storing phiX or dphiX
conv3d_blas_gpu::CpyVolConvmatImpl conv3d_blas_gpu::make_initial_CpyVolConvmatImpl(const xpuMxArrayTW &vol)
{
  CpyVolConvmatImpl ip;

  ip.vol_i.beg = 0; // to be set later
  for (int i = 0; i < 4; ++i) ip.vol_i.sz[i] = vol.getSizeAtDim(i);

  ip.convmat = this->convmat;

  if ( Y.pa_cpu != 0)
    for (int i = 0; i < 3; ++i) ip.szY[i] = this->Y.getSizeAtDim(i);
  else // dY.pa_cpu != 0
    for (int i = 0; i < 3; ++i) ip.szY[i] = this->dY.getSizeAtDim(i);

  for (int i = 0; i < 3; ++i) ip.szF[i] = this->F.getSizeAtDim(i);
  for (int i = 0; i < 3; ++i) ip.stride[i] = this->stride[i];
  for (int i = 0; i < 6; i++) ip.pad[i] = this->pad[i];

  return ip;
}

void conv3d_blas_gpu::init_convmat()
{
  // set the size
  assert( (Y.pa_cpu != 0) || (dY.pa_cpu != 0) );
  if (Y.pa_cpu != 0) // in FPROP, Y has been set
    convmat.H = numelVol(Y);
  else // (dY != 0), in BPROP, dY has been set
    convmat.H = numelVol(dY);

  convmat.W = numelVol(F) * F.getSizeAtDim(3);
  mwSize nelem = convmat.H * convmat.W;

  // allocate the memory
  void* tmp;
  hipError_t flag = hipMalloc(&tmp,  nelem*sizeof(float) ) ;
  if (flag != hipSuccess) throw conv3d_ex("Out of memory on GPU.\n");
  convmat.beg = (float*)tmp;

  // assures all zeros
  kernelSetZero<float><<<ceil_divide(nelem,NUM_THD_DIM), NUM_THD_DIM>>>(convmat.beg, nelem);
  
}

void conv3d_blas_gpu::free_convmat()
{
  hipFree( (void*)convmat.beg );
}

void conv3d_blas_gpu::vol_to_convmat (CpyVolConvmatImpl &ip, xpuMxArrayTW &vol, mwSize iInst)
{
  // set vol(:,:,:,:, i)
  ip.vol_i.beg = getVolInstDataBeg<float>(vol, iInst);

  // do the real job
  mwSize nelem = ip.convmat.H * ip.convmat.W;
  dim3 blkSize( ceil_divide(nelem, NUM_THD_DIM) );
  dim3 thdSize( NUM_THD_DIM );
  kernelCpyVolConvmat<DIR_VOL_TO_CONVMAT><<<blkSize, thdSize>>>(ip);
}

void conv3d_blas_gpu::vol_from_convmat(CpyVolConvmatImpl &ip, xpuMxArrayTW &vol, mwSize iInst)
{
  // set vol(:,:,:,:, i)
  ip.vol_i.beg = getVolInstDataBeg<float>(vol, iInst);

  // do the real job
  mwSize nelem = ip.convmat.H * ip.convmat.W;
  dim3 blkSize( ceil_divide(nelem, NUM_THD_DIM) );
  dim3 thdSize( NUM_THD_DIM );
  kernelCpyVolConvmat<DIR_VOL_FROM_CONVMAT><<<blkSize, thdSize>>>(ip);
}

void conv3d_blas_gpu::init_u()
{
  // decide the size
  assert( (Y.pa_cpu != 0) || (dY.pa_cpu != 0) );
  if (Y.pa_cpu != 0)
    u.H = numelVol(Y);
  else // (dY != 0)
    u.H = numelVol(dY);

  u.W = 1;
  mwSize nelem = u.H * u.W ;

  // allocate the memory
  void* tmp;
  hipError_t flag = hipMalloc(&tmp, nelem * sizeof(float));
  if (flag != hipSuccess) throw conv3d_ex("Out of memory on GPU.\n");
  u.beg = (float*) tmp;

  // make sure all one
  kernelSetOne<float><<<ceil_divide(nelem,NUM_THD_DIM), NUM_THD_DIM>>>(u.beg, nelem);
}

void conv3d_blas_gpu::free_u()
{
  hipFree( (void*)u.beg );
}